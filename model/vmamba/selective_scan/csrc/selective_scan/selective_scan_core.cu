/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/
#include "selective_scan_fwd_kernel.cuh"
#include "selective_scan_bwd_kernel.cuh"

template void selective_scan_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
 
template void selective_scan_bwd_cuda<float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<at::Half, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<at::BFloat16, float>(SSMParamsBwd &params, hipStream_t stream);


